
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_ERROR_CHECK

#define cudaCheckErrorDev()  __cudaCheckErrorDev(__FILE__, __LINE__)
#define cudaSafeCall(err)    __cudaSafeCall(err, __FILE__, __LINE__)
#define cudaCheckError()     __cudaCheckError(__FILE__, __LINE__)

#ifdef __cplusplus
extern "C"{
#endif 

__device__ inline void __cudaCheckErrorDev(const char *file, const int line){
  #ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if(hipSuccess != err){
      printf("%s %s %d\n", hipGetErrorString(err), file, line);
    }

    // err = cudaDeviceSynchronize(); //Ham nay chi chay duoc tren CPU (host)
    // if(cudaSuccess != err){
    //   printf("%s %s %d\n", cudaGetErrorString(err), file, line);
    // }
  #endif 
}

__host__ inline void __cudaSafeCall(hipError_t err, const char *file, const int line){
  #ifdef CUDA_ERROR_CHECK
    if(hipSuccess != err){
      fprintf(stderr, "cudaSafeCall() failed at %s:%i :%s\n", file, line, hipGetErrorString(err));
      exit(-1); 
    } 
  #endif
}               

__host__ inline void __cudaCheckError(const char *file, const int line){
  #ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if(hipSuccess != err){
      fprintf(stderr, "cudaCheckError() failed at: %s:%i: %s\n", file, line, hipGetErrorString(err));
      exit(-1);
    }
    err = hipDeviceSynchronize();
    if(hipSuccess != err){
      fprintf(stderr, "cudaCheckError() with sync failed at: %s:%i : %s\n", file, line, hipGetErrorString(err));
      exit(-1);
    }
  #endif
}

#ifdef __cplusplus
}
#endif
